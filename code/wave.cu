#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

// This aligns with the NVIDIA CUDA samples at https://github.com/NVIDIA/cuda-samples.

/*
	This example demonstrates how to use the Cuda OpenGL bindings to
	dynamically modify a vertex buffer using a Cuda kernel.

	The steps are:
	1. Create an empty vertex buffer object (VBO)
	2. Register the VBO with Cuda
	3. Map the VBO for writing from Cuda
	4. Run Cuda kernel to modify the vertex positions
	5. Unmap the VBO
	6. Render the results using OpenGL

	Host code
*/

#include <iostream>
#include <cassert>
#include <ctime>
#include <cstdlib>

#include <GL/glew.h>
#include <GL/freeglut.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <helper_timer.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

const unsigned REFRESH_DELAY = 10; //ms



////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned window_width = 512;
const unsigned window_height = 512;

const unsigned mesh_width = 256;
const unsigned mesh_height = 256;
const unsigned mesh_depth = 256;

// vbo variables
GLuint vbo;
hipGraphicsResource* cuda_vbo_resource = nullptr;
void* d_vbo_buffer = nullptr;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface* timer = nullptr;

int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
float avgFPS = 0.0f;
unsigned int frameCount = 0;

////////////////////////////////////////////////////////////////////////////////
// variables

int waveSelect;
float g_fUserAnim = 0.01f;
float meshR = 1.0f, meshG = 1.0f, meshB = 1.0f;
float userFreq = 4.0f;
int user_mesh_height = 256;
int circlePosX = mesh_width / 2;
int circlePosY = user_mesh_height / 2;
float circleRadius = 20;
int circlePosZ = (int)circleRadius * 2;

bool circleCheck = true;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool run(int argc, char** argv);
void cleanup();

// GL functionality
bool initGL(int* argc, char** argv);
void createVBO(GLuint* vbo, hipGraphicsResource** vbo_res, unsigned int vbo_res_flags);
void deleteVBO(GLuint* vbo, hipGraphicsResource* vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(hipGraphicsResource** vbo_resource);

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4* pos, unsigned int width, unsigned int height, unsigned int depth, float time, int waveSelect, float userFreq, int circlePosX, int circlePosY, int circlePosZ, float circleRadius, bool circleCheck)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	//All references to a Z coordinate were an attempt to generate a Sphere
//	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	// calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
//	float t = z / (float)depth;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
//	t = t * 2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = userFreq;
	float w[3];

	w[0] = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;
//	w[0] = sinf(u * freq + time) * cosf(v * freq + time) * sinf(t * freq * time) * 0.5f;
	w[1] = cosf(u * freq + time) * sinf(v * freq + time) * 0.5f;
	w[2] = sinf(u * freq + time) * tanf(v * freq + time) * 0.5f;
	w[3] = cosf(u * freq + time) * tanf(v * freq + time) * 0.5f;

	GLfloat circleCenterX = x - ((float)circlePosX);
	GLfloat circleCenterY = y - ((float)circlePosY);
//	GLfloat circleCenterZ = z - ((float)circlePosZ);

	if (circleCheck)
	{
		if ((circleCenterX * circleCenterX) + (circleCenterY * circleCenterY) < (circleRadius * circleRadius))
		{
			w[waveSelect] = 0;
		}
		else
		{
			w[waveSelect];
		}
	}

	//if (circleCheck)
	//{
	//	if ((circleCenterX * circleCenterX) + (circleCenterY * circleCenterY) + (circleCenterZ * circleCenterZ) < (circleRadius * circleRadius))
	//	{
	//		w[waveSelect] = 0;
	//	}
	//	else
	//	{
	//		w[waveSelect];
	//	}
	//}

	//if (waveSelect == 0)
	//{
	//	w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;
	//}
	//else if (waveSelect == 1)
	//{
	//	w = cosf(u * freq + time) * sinf(v * freq + time) * 0.5f;
	//}
	//else if (waveSelect == 2)
	//{
	//	w = sinf(u * freq + time) * tanf(v * freq + time) * 0.5f;
	//}
	//else if (waveSelect == 3)
	//{
	//	w = cosf(u * freq + time) * tanf(v * freq + time) * 0.5f;
	//}


	// write output vertex
	pos[y * width + x] = make_float4(u, w[waveSelect], v, 1.0f);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	std::cout << "wave assignment program starting...\n";

	run(argc, argv);

	std::cout << "wave assignment program completed.\n";

	return 0;
}

void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}

	char variables[256];
	sprintf(variables, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz) Mesh Height: %u Freq: %3.1f Speed: %3.1f R: %3.1f G: %3.1f B: %3.1f", avgFPS, user_mesh_height, userFreq, g_fUserAnim, meshR, meshG, meshB);
	glutSetWindowTitle(variables);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int* argc, char** argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Cuda GL Interop (VBO)");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMotionFunc(motion);
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

	glewInit();
	if (!glewIsSupported("GL_VERSION_2_0 GL_ARB_pixel_buffer_object")) {
		std::cerr << "ERROR: GL_ARB_pixel_buffer_object support missing.";
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.1, 10.0);

	assert(GL_NO_ERROR == glGetError());
	return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool run(int argc, char** argv)
{
	// Create the CUTIL timer
	sdkCreateTimer(&timer);

	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	int devID = findCudaDevice(argc, (const char**)argv);

	// First initialize OpenGL context, so we can properly set the GL for CUDA.
	// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	if (false == initGL(&argc, argv))
	{
		return false;
	}

	// register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);
	glutCloseFunc(cleanup);

	// create VBO
	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	// run the cuda part
	runCuda(&cuda_vbo_resource);

	// start rendering mainloop
	glutMainLoop();

	return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource** vbo_resource)
{

	// map OpenGL buffer object for writing from CUDA
	float4* dptr;
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes,
		*vbo_resource));
	//std::cout << "CUDA mapped VBO: May access " << num_bytes << " bytes\n";

	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	simple_vbo_kernel << <grid, block >> > (dptr, mesh_width, user_mesh_height, mesh_depth, g_fAnim, waveSelect, userFreq, circlePosX, circlePosY, circlePosZ, circleRadius, circleCheck);

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint* vbo, hipGraphicsResource** vbo_res,
	unsigned int vbo_res_flags)
{
	assert(vbo);

	// create buffer object
	glGenBuffers(1, vbo);
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);

	// initialize buffer object
	unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// register this buffer object with CUDA
	checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

	assert(GL_NO_ERROR == glGetError());
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res)
{
	// unregister this buffer object with CUDA
	checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
	sdkStartTimer(&timer);

	// run CUDA kernel to generate vertex positions
	runCuda(&cuda_vbo_resource);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, translate_z);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);

	// render from the vbo
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(4, GL_FLOAT, 0, 0);

	glEnableClientState(GL_VERTEX_ARRAY);
	glColor3f(meshR, meshG, meshB);
	glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
	glDisableClientState(GL_VERTEX_ARRAY);

	glutSwapBuffers();

	g_fAnim += g_fUserAnim;

	sdkStopTimer(&timer);
	computeFPS();
}

void timerEvent(int value)
{
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

	}
}

void cleanup()
{
	sdkDeleteTimer(&timer);

	if (vbo)
	{
		deleteVBO(&vbo, cuda_vbo_resource);
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case (27):
		glutDestroyWindow(glutGetWindow());
	case ('.'):
		if (g_fUserAnim < 0.10f)
			g_fUserAnim += 0.01f;
		break;
	case (','):
		if (g_fUserAnim >= 0.01f)
			g_fUserAnim -= 0.01f;
		if (g_fUserAnim < 0.01f)
			g_fUserAnim = 0.01f;
		break;
	case ('p'):
		if (meshR < 1.0f)
			meshR += 0.1f;
		break;
	case ('o'):
		if (meshR >= 0.0f)
			meshR -= 0.1f;
		break;
	case ('l'):
		if (meshG < 1.0f)
			meshG += 0.1f;
		break;
	case ('k'):
		if (meshG >= 0.0f)
			meshG -= 0.1f;
		break;
	case ('m'):
		if (meshB < 1.0f)
			meshB += 0.1f;
		break;
	case ('n'):
		if (meshB >= 0.0f)
			meshB -= 0.1f;
		break;
	case ('x'):
		if (userFreq < 10.0f)
			userFreq += 1.0f;
		break;
	case ('z'):
		if (userFreq >= 1.0f)
			userFreq -= 1.0f;
		if (userFreq < 1.0f)
			userFreq = 1.0f;
		break;
	case ('c'):
		if (user_mesh_height >= 1)
			user_mesh_height -= 1;
		if (user_mesh_height < 1)
			user_mesh_height = 1;
		break;
	case ('v'):
		if (user_mesh_height < 1024)
			user_mesh_height += 1;
		break;
	case ('w'):
		circlePosY++;
		break;
	case ('s'):
		circlePosY--;
		break;
	case ('d'):
		circlePosX++;
		break;
	case ('a'):
		circlePosX--;
		break;
	case ('1'):
		waveSelect = 0;
		break;
	case ('2'):
		waveSelect = 1;
		break;
	case ('3'):
		waveSelect = 2;
		break;
	case ('4'):
		waveSelect = 3;
		break;
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1 << button;
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

void motion(int x, int y)
{
	float dx, dy;
	dx = (float)(x - mouse_old_x);
	dy = (float)(y - mouse_old_y);

	if (mouse_buttons & 1)
	{
		rotate_x += dy * 0.2f;
		rotate_y += dx * 0.2f;
	}
	else if (mouse_buttons & 4)
	{
		translate_z += dy * 0.01f;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}